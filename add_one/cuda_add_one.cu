#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addOne(int *a, int n) {
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadID < n) {
        a[threadID] += 1;
    }
}

void printArray(int *a, int n) {
    for (int i = 0; i < n; ++i) {
        std::cout << "Element at index " << i << ": " << a[i] << std::endl;
    }
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0); // 获取第0个设备的信息
        std::cout << "Running on GPU: " << prop.name << std::endl;
    }

    const long long int N = 1000000000; // 调整数组大小到一个合理的值，以免超出GPU显存限制
    int *h_a, *d_a;

    // 在主机上分配和初始化内存
    h_a = new int[N];
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
    }

    // 在设备（GPU）上分配内存
    hipMalloc((void**)&d_a, N * sizeof(int));

    // 将主机数据复制到设备
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);

    // 循环启动CUDA内核函数
    for (int i = 0; i < 10000; ++i) {
        // 启动CUDA内核函数
        addOne<<<1, N>>>(d_a, N);
    }

    // 确保所有CUDA kernel都已完成执行
    hipDeviceSynchronize();

    // 将设备上的结果复制回主机
    hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);

    // 打印主机上的结果
    printArray(h_a, N);

    // 释放内存
    hipFree(d_a);
    delete[] h_a;

    return 0;
}
